#include "hip/hip_runtime.h"
#include<thrust/device_vector.h>
#include<thrust/host_vector.h>
#include <thrust/async/for_each.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <iostream>
#include <chrono>

#define TOL 0.000001

int main(int argc, char **argv)
{
    std::cout << argv[0] << std::endl;
    
    int N=atoi(argv[1]);
    float A{2.0};
 
    thrust::device_vector<float> X(N);
    thrust::device_vector<float> Y(N);
    thrust::device_vector<float> Z(N);

    auto t1 = std::chrono::steady_clock::now();   // Start timing     
         
    auto e1=thrust::async::for_each(
        thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(0),X.begin(),Y.begin())),
        thrust::make_zip_iterator(thrust::make_tuple(thrust::make_counting_iterator(N),X.end(),Y.end())),
        [=]__device__ (auto tup)
        {
            int i=thrust::get<0>(tup);
            thrust::get<1>(tup)=1.0/float(i+1);
            thrust::get<2>(tup)=-2.0/float(i+1);            
        }
    );    
      
    auto e2=thrust::async::for_each(
    thrust::make_zip_iterator(thrust::make_tuple(X.begin(),Y.begin(),Z.begin())),
    thrust::make_zip_iterator(thrust::make_tuple(X.end(),Y.end(),Z.end())),
        [=]__device__ (auto tup)
        {
            thrust::get<2>(tup)=A*thrust::get<0>(tup)+thrust::get<1>(tup);    
        }
    );
   
   auto t2 = std::chrono::steady_clock::now();   // Start timing
   hipDeviceSynchronize();   
   auto t3 = std::chrono::steady_clock::now();   // Start timing

   thrust::host_vector<float> Xh(X);
   thrust::host_vector<float> Yh(Y);
   thrust::host_vector<float> Zh(Z);

   auto t4 = std::chrono::steady_clock::now();   // Start timing

    // check
    for (int i = 0; i < N; i++) {
    assert(fabs(A * Xh[i] + Yh[i] - Zh[i]) < TOL);
    }

    std::cout
    << "before Q.wait "
    << std::chrono::duration_cast<std::chrono::microseconds>(t2 - t1).count()
    << std::endl;
    
    std::cout
    << "after Q.wait "
    << std::chrono::duration_cast<std::chrono::microseconds>(t3 - t1).count()
    << std::endl;

    std::cout
      << "after device to host copy "
      << std::chrono::duration_cast<std::chrono::microseconds>(t4 - t1).count()
      << std::endl;
}